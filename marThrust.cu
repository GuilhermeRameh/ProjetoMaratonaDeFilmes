#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <iostream>
#include <vector>

using namespace std;

struct filme
{
    int id;
    int hInicio;
    int hFinal;
    int categoria;
};


struct AddFunctor {
    __host__ __device__
    int operator()(int i, int ncategorias, int nfilmes) {

        bitset<24> horas(0x0000000000000000000);
        bitset<64> bitsetFilmes(0);
        vector<int> localfilmesPorCats;
        localfilmesPorCats.resize(ncategorias);

        int local_total_iteration = 0;
        bitsetFilmes = i;
        horas = 0;

        for (int i=0; i<ncategorias; i++)
        {
            localfilmesPorCats[i] = fpcReset[i] ;
        }

        // cout << omp_get_thread_num() << " " << i << endl;

        for (int j=0; j<nfilmes; j++)
        {            
            if (bitsetFilmes[j])
            {
                filme valor = filmes[j];
                
                if (valor.hInicio <= valor.hFinal){
                    if (localfilmesPorCats[valor.categoria] <= 0)
                    {
                        break;
                    }
                    bitset<24> mascara;
                    for (int i = valor.hInicio; i <= valor.hFinal; i++) {
                        mascara.set(i);
                    }

                    // cout << "\nInicio: " << valor.hInicio << " Final: " << valor.hFinal;

                    bitset<24> resultado = horas & mascara;

                    if (resultado == 0)
                    {
                        for (int i = valor.hInicio; i <= valor.hFinal; i++) {
                            horas.set(i);
                        }
                        localfilmesPorCats[valor.categoria]--;
                        local_total_iteration += 1;
                        // cout << "  > Iter - " << i << "   ID: " << valor.id << endl;
                    }
                }
            }
        }


        return x;
    }
};


void ParTHRUST()
{
    int nfilmes, ncategorias, comp_total = 0;
    cin >> nfilmes >> ncategorias;

    vector<int> filmesPorCats, fpcReset, total_values;
    vector<filme> filmes;

    filmesPorCats.resize(ncategorias);
    fpcReset.resize(ncategorias);

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<ncategorias; i++)
    {
        cin >> filmesPorCats[i];
        fpcReset[i] = filmesPorCats[i];
    }

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<nfilmes; i++)
    {
        filme this_film;
        this_film.id = i;
        cin >> this_film.hInicio >> this_film.hFinal >> this_film.categoria;
        filmes.push_back(this_film);
    }

    int nExpo = pow(2, nfilmes);


    thrust::device_vector<int> input(nExpo);
    for (int i = 0; i < nExpo; i++)
    {
        input[i] = i;
    }



    std::cout << "\nTOTAL GLOBAL: " << comp_total << std::endl;
}


int main()
{
    ParTHRUST();
    return 0;
}
#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <iostream>
#include <vector>
#include <cmath>

using namespace std;

struct filme
{
    int id;
    int hInicio;
    int hFinal;
    int categoria;
};


struct CheckFilm {
    int ncategorias;
    int nfilmes;
    thrust::host_vector<int> fpcReset;
    thrust::host_vector<filme> filmes;

    CheckFilm(int ncat, int nflm, thrust::host_vector<int> reset, thrust::host_vector<filme> flm) : 
    ncategorias(ncat), nfilmes(nflm), fpcReset(reset), filmes(flm) {}

    __host__ __device__
    int operator()(int i)
    {
        thrust::device_vector<bool> horas(24);
        thrust::device_vector<bool> bitsetFilmes(64);
        thrust::device_vector<int> localfilmesPorCats;
        localfilmesPorCats.resize(ncategorias);

        int local_total_iteration = 0;
        int temp_num = nfilmes;
        
        // Prepara o vetor de filmes
        while (temp_num > 0)
        {
            bitsetFilmes.push_back(temp_num & 1);
            temp_num >>= 1;
        }

        // Reseta as Horas
        for (int j=0; j<24; j++)
        {
            horas[j] = 0;
        }
        

        for (int i=0; i<ncategorias; i++)
        {
            localfilmesPorCats[i] = fpcReset[i] ;
        }

        // cout << omp_get_thread_num() << " " << i << endl;

        for (int j=0; j<nfilmes; j++)
        {            
            if (bitsetFilmes[j])
            {
                filme valor = filmes[j];
                
                if (valor.hInicio <= valor.hFinal){
                    if (localfilmesPorCats[valor.categoria] <= 0)
                    {
                        break;
                    }
                    thrust::device_vector<bool> mascara(24);
                    for (int i = valor.hInicio; i <= valor.hFinal; i++) {
                        mascara[i] = true;
                    }

                    // cout << "\nInicio: " << valor.hInicio << " Final: " << valor.hFinal;

                    bool resultado;
                    for (int i=0; i<24; i++)
                    {
                        resultado &= horas[i] & mascara[i];
                    }

                    if (resultado == false)
                    {
                        for (int i = valor.hInicio; i <= valor.hFinal; i++) {
                            horas[i] = true;
                        }
                        localfilmesPorCats[valor.categoria]--;
                        local_total_iteration += 1;
                        // cout << "  > Iter - " << i << "   ID: " << valor.id << endl;
                    }
                }
            }
        }
        return local_total_iteration;
    }    
};


void ParTHRUST()
{   
    int nfilmes, ncategorias, comp_total = 0;
    cin >> nfilmes >> ncategorias;

    vector<int> filmesPorCats, fpcReset, total_values;
    vector<filme> filmes;

    filmesPorCats.resize(ncategorias);
    fpcReset.resize(ncategorias);

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<ncategorias; i++)
    {
        cin >> filmesPorCats[i];
        fpcReset[i] = filmesPorCats[i];
    }

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<nfilmes; i++)
    {
        filme this_film;
        this_film.id = i;
        cin >> this_film.hInicio >> this_film.hFinal >> this_film.categoria;
        filmes.push_back(this_film);
    }

    int nExpo = pow(2, nfilmes);


    thrust::device_vector<int> input(nExpo);
    thrust::device_vector<int> output(nExpo);
    for (int i = 0; i < nExpo; i++)
    {
        input[i] = i;        
    }

    CheckFilm myFunctor(ncategorias, nfilmes, fpcReset, filmes);

    thrust::transform(input.begin(), input.end(), output.begin(), myFunctor);

    std::cout << "\nTOTAL GLOBAL: " << comp_total << std::endl;
}


int main(void)
{
    ParTHRUST();
    return 0;
}
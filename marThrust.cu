#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <iostream>
#include <vector>
#include <cmath>

using namespace std;

struct filme
{
    int id;
    int hInicio;
    int hFinal;
    int categoria;
};


void ParTHRUST()
{   
    int nfilmes, ncategorias, comp_total = 0;
    cin >> nfilmes >> ncategorias;

    thrust::host_vector<int> filmesPorCats, fpcReset, total_values;
    thrust::host_vector<filme> filmes;

    filmesPorCats.resize(ncategorias);
    fpcReset.resize(ncategorias);

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<ncategorias; i++)
    {
        cin >> filmesPorCats[i];
        fpcReset[i] = filmesPorCats[i];
    }

    // NOTE: Não paralelizavel pois possui o CIN
    for (int i=0; i<nfilmes; i++)
    {
        filme this_film;
        this_film.id = i;
        cin >> this_film.hInicio >> this_film.hFinal >> this_film.categoria;
        filmes.push_back(this_film);
    }

    int nExpo = pow(2, nfilmes);

    thrust::device_vector<int> t_fpcReset = fpcReset;
    thrust::device_vector<filme> t_filmes = filmes;

    thrust::device_vector<int> output(nExpo);
    for (int i = 0; i < nExpo; i++)
    {
        thrust::device_vector<bool> horas(24, false);
        thrust::device_vector<bool> bitsetFilmes(nfilmes, false);
        thrust::device_vector<int> localfilmesPorCats;
        localfilmesPorCats.resize(ncategorias);

        int local_total_iteration = 0;
        int temp_num = i;
        int iter = 0;
        
        // Prepara o vetor de filmes
        while (temp_num > 0)
        {
            bitsetFilmes[iter] = (temp_num & 1);
            temp_num >>= 1;
            iter++;
        }
       
       
        // Reseta as Horas
        for (int j=0; j<24; j++)
        {
            horas[j] = false;
        }
        

        for (int j = 0; j < ncategorias; j++)
        {
            localfilmesPorCats[j] = t_fpcReset[j];
        }

        // cout << omp_get_thread_num() << " " << i << endl;

        for (int j=0; j<nfilmes; j++)
        {            
            if (bitsetFilmes[j])
            {
                filme valor = t_filmes[j];

                if (valor.hInicio <= valor.hFinal){
                    if (localfilmesPorCats[valor.categoria-1] <= 0)
                    {
                        break;
                    }
                    thrust::device_vector<bool> mascara(24);
                    for (int k = valor.hInicio; k <= valor.hFinal; k++) {
                        mascara[k] = true;
                    }

                    // cout << "\nInicio: " << valor.hInicio << " Final: " << valor.hFinal;

                    bool resultado = true;
                    for (int k=0; k<24; k++)
                    {
                        resultado = resultado && horas[k] && mascara[k];
                    }

                    if (resultado == false)
                    {
                        for (int k = valor.hInicio; k <= valor.hFinal; k++) {
                            horas[k] = true;
                        }
                        localfilmesPorCats[valor.categoria-1] = localfilmesPorCats[valor.categoria-1]-1;
                        local_total_iteration += 1;
                    }
                }
            }
        }
        output[i] = local_total_iteration;
    }


    comp_total = thrust::reduce(output.begin(), output.end(), -1, thrust::maximum<int>());

    std::cout << "\nTOTAL GLOBAL: " << comp_total << std::endl;
}


int main(void)
{
    ParTHRUST();
    return 0;
}

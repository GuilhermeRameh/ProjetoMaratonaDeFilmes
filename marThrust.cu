#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <iostream>
#include <vector>
#include <cmath>

using namespace std;

struct filme
{
    int id;
    int hInicio;
    int hFinal;
    int categoria;
};


// void ParTHRUST()
// {
//     __device__

//     int nfilmes, ncategorias, comp_total = 0;
//     cin >> nfilmes >> ncategorias;

//     vector<int> filmesPorCats, fpcReset, total_values;
//     vector<filme> filmes;

//     filmesPorCats.resize(ncategorias);
//     fpcReset.resize(ncategorias);

//     // NOTE: Não paralelizavel pois possui o CIN
//     for (int i=0; i<ncategorias; i++)
//     {
//         cin >> filmesPorCats[i];
//         fpcReset[i] = filmesPorCats[i];
//     }

//     // NOTE: Não paralelizavel pois possui o CIN
//     for (int i=0; i<nfilmes; i++)
//     {
//         filme this_film;
//         this_film.id = i;
//         cin >> this_film.hInicio >> this_film.hFinal >> this_film.categoria;
//         filmes.push_back(this_film);
//     }

//     int nExpo = pow(2, nfilmes);


//     thrust::device_vector<int> output(nExpo);
//     for (int i = 0; i < nExpo; i++)
//     {
        
//         vector<bool> horas(24);
//         vector<bool> bitsetFilmes(64);
//         vector<int> localfilmesPorCats;
//         localfilmesPorCats.resize(ncategorias);

//         int local_total_iteration = 0;
//         int temp_num = nfilmes;
        
//         // Prepara o vetor de filmes
//         while (temp_num > 0)
//         {
//             bitsetFilmes.push_back(temp_num & 1);
//             cout << (temp_num & 1);
//             temp_num >>= 1;
//         }

//         // Reseta as Horas
//         for (int j=0; j<24; j++)
//         {
//             horas[j] = 0;
//         }
        

//         for (int i=0; i<ncategorias; i++)
//         {
//             localfilmesPorCats[i] = fpcReset[i] ;
//         }

//         // cout << omp_get_thread_num() << " " << i << endl;

//         for (int j=0; j<nfilmes; j++)
//         {            
//             if (bitsetFilmes[j])
//             {
//                 filme valor = filmes[j];
                
//                 if (valor.hInicio <= valor.hFinal){
//                     if (localfilmesPorCats[valor.categoria] <= 0)
//                     {
//                         break;
//                     }
//                     vector<bool> mascara(24);
//                     for (int i = valor.hInicio; i <= valor.hFinal; i++) {
//                         mascara[i] = true;
//                     }

//                     // cout << "\nInicio: " << valor.hInicio << " Final: " << valor.hFinal;

//                     bool resultado;
//                     for (int i=0; i<24; i++)
//                     {
//                         resultado &= horas[i] & mascara[i];
//                     }

//                     if (resultado == 0)
//                     {
//                         for (int i = valor.hInicio; i <= valor.hFinal; i++) {
//                             horas[i] = true;
//                         }
//                         localfilmesPorCats[valor.categoria]--;
//                         local_total_iteration += 1;
//                         // cout << "  > Iter - " << i << "   ID: " << valor.id << endl;
//                     }
//                 }
//             }
//         }
//     }

//     std::cout << "\nTOTAL GLOBAL: " << comp_total << std::endl;
// }


int main(void)
{
    // ParTHRUST();

    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;
    
    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for(size_t i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);
    
    std::cout << "H now has size " << H.size() << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;
    
    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;
    
    // print contents of D
    for(size_t i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns
    return 0;
}